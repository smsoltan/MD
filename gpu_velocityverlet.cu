#include "hip/hip_runtime.h"
//nvcc -o gpu_velocityverlet gpu_velocityverlet.cu
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <hip/hip_vector_types.h>
#include "book.h"
//#include "cuPrintf.cu"

#define SIGMA 1
#define RC 2.5*SIGMA
#define EPS 0.01
#define ZERO 4*EPS*(pow(SIGMA/RC, 12) - pow(SIGMA/RC, 6))

//double OFF = 4*EPS*(pow(2.5, -12) - powf(2.5, -6))

int N, l;
double S, step;

__constant__ int N_dev, l_dev;
__constant__ double S_dev;
__constant__ double step_dev;

// these exist on the gpu side
//texture<double> tex_r, tex_rn;


int blocksPerGrid;
const int threadsPerBlock = 256;

//GPU lock-free synchronization function
//modification: volatile it is.
//modification: now it also finishes summing K and V
__device__ void __gpu_sync(int goalVal, volatile int *Arrayin,
                                        volatile int *Arrayout,
                           float mK, float mV,
                           float *Kb, float *Vb,
                           float *K, float *V)
{
 // thread ID in a block
 int tid_in_block = threadIdx.x * blockDim.y + threadIdx.y;
 int nBlockNum = gridDim.x * gridDim.y;
 int bid = blockIdx.x * gridDim.y + blockIdx.y;
 // only thread 0 is used for synchronization
 if (tid_in_block == 0)
 {
  Kb[bid] = mK; Vb[bid] = mV;
  Arrayin[bid] = goalVal;
 }
 if (bid == 1)
 {
  if (tid_in_block < nBlockNum)
  {
    while (Arrayin[tid_in_block] != goalVal);
  }
  __shared__ float2 cache[threadsPerBlock];
  if (tid_in_block < nBlockNum) {cache[tid_in_block].x = Kb[tid_in_block];
                                 cache[tid_in_block].y = Vb[tid_in_block];}
  int i = nBlockNum/2;
  __syncthreads();
  while(i != 0)
  {
   if(tid_in_block < i) {cache[tid_in_block].x += cache[tid_in_block+i].x;
                         cache[tid_in_block].y += cache[tid_in_block+i].y;}
   __syncthreads();
   i /= 2;
  }
  if (tid_in_block == 0) {K[goalVal] = cache[0].x; V[goalVal] = cache[0].y;}
  if (tid_in_block < nBlockNum)
  {
   Arrayout[tid_in_block] = goalVal;
  }
 }
 if (tid_in_block == 0)
 {
  while (Arrayout[bid] != goalVal);
 }
 __syncthreads();
}


/*it seems, that if we want one kernel, we need to calculate r at the end. But
 * since we need r from the next step to eventually calculate v for this next
 * step, we need the initialising kernel*/

__global__ void InitVelocityVerlet
           (double *r, double *v, double *a, double *rn)
{
   int tid = threadIdx.x + blockIdx.x * blockDim.x;
//   if (tid == 0) locke = 0;

//calculating a
   double ax_new = 0, ay_new = 0, az_new = 0,
         x_old = r[3*tid],//tex1Dfetch(tex_r, 3*tid),
         y_old = r[3*tid+1],//tex1Dfetch(tex_r, 3*tid+1),
         z_old = r[3*tid+2];//tex1Dfetch(tex_r, 3*tid+2);
   for(int j = 0; j < N_dev; ++j)
   {
      if (tid != j)
      {
         double x = x_old - r[3*j];//tex1Dfetch(tex_r, 3*j);
         if (x > RC) x-= S_dev; else if (x < -RC) x+= S_dev;
         double y = y_old - r[3*j+1];//tex1Dfetch(tex_r, 3*j+1);
         if (y > RC) y-= S_dev; else if (y < -RC) y+= S_dev;
         double z = z_old - r[3*j+2];//tex1Dfetch(tex_r, 3*j+2);
         if (z > RC) z-= S_dev; else if (z < -RC) z+= S_dev;
         double r2 = x*x + y*y + z*z;
         if (r2 < RC*RC)
         {
            double R = pow(SIGMA*SIGMA/r2, 3);
            double part = 4*EPS*R*R, ULJ = part - 4*EPS*R;
            ax_new += 6*x*(part + ULJ)/r2;
            ay_new += 6*y*(part + ULJ)/r2;
            az_new += 6*z*(part + ULJ)/r2;
         }
      }
   }
   a[3*tid] = ax_new; a[3*tid+1] = ay_new; a[3*tid+2] = az_new;

//calculating r
   double x_new = x_old 
                 + step_dev*v[3*tid]//tex1Dfetch(tex_v, 3*tid)
                 + 0.5*ax_new*step_dev*step_dev;
   if (x_new > S_dev) {x_new -= S_dev; }
   else if (x_new < 0) {x_new += S_dev; }
   double y_new = y_old
                 + step_dev*v[3*tid+1]//tex1Dfetch(tex_v, 3*tid+1)
                 + 0.5*ay_new*step_dev*step_dev;
   if (y_new > S_dev) {y_new -= S_dev; }
   else if (y_new < 0) {y_new += S_dev; }
   double z_new = z_old
                 + step_dev*v[3*tid+2]//tex1Dfetch(tex_v, 3*tid+2)
                 + 0.5*az_new*step_dev*step_dev;
   if (z_new > S_dev) {z_new -= S_dev; }
   else if (z_new < 0) {z_new += S_dev; }
   rn[3*tid] = x_new;
   rn[3*tid+1] = y_new;
   rn[3*tid+2] = z_new;

}

__device__ float acalc(double *ax_new, double *ay_new, double *az_new,
     volatile double *x_load, volatile double *y_load, volatile double *z_load,
     volatile double *ro)
{
   int cacheIndex = threadIdx.x;
   int tid = threadIdx.x + blockIdx.x * blockDim.x;
   double sum = 0;
//   ax_new = 0; ay_new = 0; az_new = 0;
   for(int ii = blockIdx.x + 1; ii < blockIdx.x+gridDim.x; ++ii)
   {
      int i = (ii%gridDim.x)*blockDim.x;
      x_load[cacheIndex+i] = ro[3*(cacheIndex+i)];
      y_load[cacheIndex+i] = ro[3*(cacheIndex+i)+1];
      z_load[cacheIndex+i] = ro[3*(cacheIndex+i)+2];
   }
      __syncthreads();
      for(int j = 0; j < N_dev; ++j)
         if(j != tid)
         {
            double x, y, z;
            x = x_load[tid] - x_load[j];//tex1Dfetch(tex_rn, 3*j);
            y = y_load[tid] - y_load[j];//tex1Dfetch(tex_rn, 3*j+1);
            z = z_load[tid] - z_load[j];//tex1Dfetch(tex_rn, 3*j+2);}
            if (x > RC) x-= S_dev; else if (x < -RC) x+= S_dev;
            if (y > RC) y-= S_dev; else if (y < -RC) y+= S_dev;
            if (z > RC) z-= S_dev; else if (z < -RC) z+= S_dev;
            double r2 = x*x + y*y + z*z;
            if (r2 < RC*RC)
            {
               double R = pow(SIGMA*SIGMA/r2, 3);
               double part = 4*EPS*R*R, ULJ = part - 4*EPS*R;
               *ax_new += 6*x*(part + ULJ)/r2;
               *ay_new += 6*y*(part + ULJ)/r2;
               *az_new += 6*z*(part + ULJ)/r2;
               sum += ULJ - ZERO;
            }

         }
//      for(int j = tid+1; j < N_dev; ++j)
//         sum += subacalc(j, ax_new, ay_new, az_new, x_load, y_load, z_load);
   return static_cast<float>(sum);
}

//This version calls a thread for every particle.
__global__ void VelocityVerlet
                (double *r, double *v, double *a, 
                 double *rn, float *Kb, float *Vb,
                 float *K, float *V,
                 int *Arrayin, int *Arrayout)
{
   int tid = threadIdx.x + blockIdx.x * blockDim.x;
   int cacheIndex = threadIdx.x;
//   volatile int bid = blockIdx.x;
   __shared__ float cache[threadsPerBlock];
   __shared__ float cache2[threadsPerBlock];
//   __shared__ double cache3[threadsPerBlock];
   extern __shared__ double array[];
   volatile double *x_load = array;
   volatile double *y_load = &x_load[N_dev];
   volatile double *z_load = &y_load[N_dev];
   
   if (cacheIndex == 0) Arrayin[blockIdx.x] = -1;
   else if (cacheIndex == 1) Arrayout[blockIdx.x] = -1;

//   bool whichTexture = true;
   double *temp = r; r = rn; rn = temp;
   volatile double ax_old = a[3*tid],//tex1Dfetch(tex_a, 3*tid),
         ay_old = a[3*tid+1],//tex1Dfetch(tex_a, 3*tid+1),
         az_old = a[3*tid+2];//tex1Dfetch(tex_a, 3*tid+2);
   double vx = v[3*tid],
          vy = v[3*tid+1],
          vz = v[3*tid+2];
   int old = cacheIndex + blockIdx.x*blockDim.x;
   x_load[old] = r[3*tid];//tex1Dfetch(tex_r, 3*tid),
   y_load[old] = r[3*tid+1],//tex1Dfetch(tex_r, 3*tid+1),
   z_load[old] = r[3*tid+2];//tex1Dfetch(tex_r, 3*tid+2);

for (int it = 0; it < l_dev; ++it)
{
//calculating a
   double ax_new = 0, ay_new = 0, az_new = 0; 
   cache2[cacheIndex] = acalc(&ax_new, &ay_new, &az_new,
                              x_load, y_load, z_load, r);

//calculating v
   vx = vx//tex1Dfetch(tex_v, 3*tid)
        + 0.5*step_dev*(ax_old + ax_new),
   vy = vy//tex1Dfetch(tex_v, 3*tid + 1)
        + 0.5*step_dev*(ay_old + ay_new),
   vz = vz//tex1Dfetch(tex_v, 3*tid + 2)
        + 0.5*step_dev*(az_old + az_new);
//   v[3*tid] = vx; v[3*tid+1] = vy; v[3*tid+2] = vz;

//measurement of temperature
   cache[cacheIndex] = static_cast<float>(vx*vx + vy*vy + vz*vz);
   __syncthreads();
   int i = blockDim.x/2;
   while (i != 0)
   {
      if (cacheIndex < i)
      {
         cache[cacheIndex] += cache[cacheIndex + i];
         cache2[cacheIndex] += cache2[(cacheIndex+i)];
      }
      __syncthreads();
      i /= 2;
   }

/*   if (cacheIndex == 0)
      K[it*gridDim.x + blockIdx.x] = cache[0]/2;
   else if (cacheIndex == 1)
      V[it*gridDim.x + blockIdx.x] = cache2[0]/2;*/

//calculating r
   double x_new = x_load[old]
                 + vx*step_dev
                 + 0.5*ax_new*step_dev*step_dev;
   if (x_new > S_dev) {x_new -= S_dev; }
   else if (x_new < 0) {x_new += S_dev; }
   double y_new = y_load[old]
                 + vy*step_dev
                 + 0.5*ay_new*step_dev*step_dev;
   if (y_new > S_dev) {y_new -= S_dev; }
   else if (y_new < 0) {y_new += S_dev; }
   double z_new = z_load[old]
                 + vz*step_dev
                 + 0.5*az_new*step_dev*step_dev;
   if (z_new > S_dev) {z_new -= S_dev; }
   else if (z_new < 0) {z_new += S_dev; }
   rn[3*tid] = x_new;
   rn[3*tid+1] = y_new;
   rn[3*tid+2] = z_new;

   temp = r; r = rn; rn = temp;
//   whichTexture = !whichTexture;
   ax_old = ax_new; ay_old = ay_new; az_old = az_new;
   x_load[old] = x_new; y_load[old] = y_new; z_load[old] = z_new;
   __gpu_sync(it, Arrayin, Arrayout,
              cache[0]/2, cache2[0]/2, Kb, Vb, K, V);
}
}

int setAtoms(int n, double b, double *r)
{
   S = n*b;
   if (S < 3*RC) {l = 0; return 1; }
   for (int z = 0; z < n; ++z)
      for (int y = 0; y < n; ++y)
         for (int x = 0; x < n; ++x)
   {
      r[3*(z*n*n + y*n + x)] = ((double) x)*b + 0.5*b;
      r[3*(z*n*n + y*n + x) + 1] = y*b + 0.5*b;
      r[3*(z*n*n + y*n + x) + 2] = z*b + 0.5*b;
   }
   return 0;
}
void Write(float *K, float *V, FILE *ofp)
{
   //sumall(obs);
   for(int i = 0; i < l; ++i)
   {
      double t = i*step;
      double a = K[i], b = V[i];
      fprintf(ofp, "%f %f %f %f\n", 
                   t, a, b, a+b);
   }

}



int main(int argc, char **argv)
{
   char *iname = argv[1], *oname = argv[2];
   double b = atof(argv[3]);
   step = atof(argv[4]);
   l = atoi(argv[5]);

   FILE *fp; fp = fopen(iname, "r");
   int n;
   fscanf(fp, "%i", &n);
   N = n*n*n;
   double *r = (double*) malloc(sizeof(double) * 3*N), 
         *v = (double*) malloc(sizeof(double) * 3*N);
   setAtoms(n, b, r);
   blocksPerGrid = N / threadsPerBlock;
   for(int i = 0; i < 3*N; ++i)
   {
      float vi;
      if (fscanf(fp, "%f", &vi) == EOF) l = 0;
      v[i] = static_cast<double>(vi);
   }
   fclose(fp);
   hipEvent_t start, stop;
   HANDLE_ERROR( hipEventCreate( &start ) );
   HANDLE_ERROR( hipEventCreate( &stop ) );
   HANDLE_ERROR( hipEventRecord( start, 0 ) );

   hipMemcpyToSymbol(HIP_SYMBOL(S_dev),
                   &S,
                   1*sizeof(double),
                   0,
                   hipMemcpyHostToDevice);
   hipMemcpyToSymbol(HIP_SYMBOL(step_dev),
                   &step,
                   1*sizeof(double),
                   0,
                   hipMemcpyHostToDevice);
   hipMemcpyToSymbol(HIP_SYMBOL(N_dev),
                   &N,
                   1*sizeof(int),
                   0,
                   hipMemcpyHostToDevice);
   hipMemcpyToSymbol(HIP_SYMBOL(l_dev),
                   &l,
                   1*sizeof(int),
                   0,
                   hipMemcpyHostToDevice);

   double *dev_r, *dev_v, *dev_a, *dev_rn /**dev_an*/;
   HANDLE_ERROR( hipMalloc( (void**)&dev_r,
                              3*N*sizeof(double) ) );
   HANDLE_ERROR( hipMalloc( (void**)&dev_v,
                              3*N*sizeof(double) ) );
   HANDLE_ERROR( hipMalloc( (void**)&dev_a,
                              3*N*sizeof(double) ) );
   HANDLE_ERROR( hipMalloc( (void**)&dev_rn,
                              3*N*sizeof(double) ) );
//   HANDLE_ERROR( hipBindTexture( NULL, tex_r,
//                                  dev_r, 3*N*sizeof(float)));
//   HANDLE_ERROR( hipBindTexture( NULL, tex_rn,
//                                  dev_rn, 3*N*sizeof(float)));
   
   HANDLE_ERROR( hipMemcpy( dev_r, r, 3*N*sizeof(double),
                              hipMemcpyHostToDevice ) );
   HANDLE_ERROR( hipMemcpy( dev_v, v, 3*N*sizeof(double),
                              hipMemcpyHostToDevice ) );
   
   float *dev_Kb, *dev_Vb;
   HANDLE_ERROR( hipMalloc( (void**)&dev_Kb,
                              blocksPerGrid*sizeof(float) ) );
   HANDLE_ERROR( hipMalloc( (void**)&dev_Vb,
                              blocksPerGrid*sizeof(float) ) );
   float *dev_K, *dev_V;
   HANDLE_ERROR( hipMalloc( (void**)&dev_K,
                              l*sizeof(float) ) );
   HANDLE_ERROR( hipMalloc( (void**)&dev_V,
                              l*sizeof(float) ) );
   int *dev_Arrayin, *dev_Arrayout;
   HANDLE_ERROR( hipMalloc( (void**)&dev_Arrayin,
                              blocksPerGrid*sizeof(int) ) );
   HANDLE_ERROR( hipMalloc( (void**)&dev_Arrayout,
                              blocksPerGrid*sizeof(int) ) );
   
//   cudaPrintfInit();
   InitVelocityVerlet<<<blocksPerGrid, threadsPerBlock>>>
                    (dev_r, dev_v, dev_a, dev_rn);

   VelocityVerlet<<<blocksPerGrid, threadsPerBlock, 3*N*sizeof(double)>>>
                (dev_r, dev_v, dev_a, dev_rn,
                 dev_Kb, dev_Vb, dev_K, dev_V,
                 dev_Arrayin, dev_Arrayout);
   
/*   hipFree(dev_r);
   hipFree(dev_v);
   hipFree(dev_a);
   hipFree(dev_rn);
   double *dev_K, *dev_V;
   HANDLE_ERROR( hipMalloc( (void**)&dev_K,
                              l*sizeof(double) ) );
   HANDLE_ERROR( hipMalloc( (void**)&dev_V,
                              l*sizeof(double) ) );
   Finish<<<l, blocksPerGrid>>>
        (dev_Kb, dev_Vb, dev_K, dev_V);*/
   float *K = (float*) malloc(sizeof(float) * l),
         *V = (float*) malloc(sizeof(float) * l);
   HANDLE_ERROR( hipMemcpy( K, dev_K, l*sizeof(float),
                              hipMemcpyDeviceToHost ) );
   HANDLE_ERROR( hipMemcpy( V, dev_V, l*sizeof(float),
                              hipMemcpyDeviceToHost ) );
//   cudaPrintfDisplay();
//   hipUnbindTexture(tex_r);
//   hipUnbindTexture(tex_rn);

   hipFree(dev_r);
   hipFree(dev_v);
   hipFree(dev_a);
   hipFree(dev_rn);

   hipFree(dev_Kb);
   hipFree(dev_Vb);
   hipFree(dev_K);
   hipFree(dev_V);
   hipFree(dev_Arrayin);
   hipFree(dev_Arrayout);

   HANDLE_ERROR( hipEventRecord( stop, 0 ) );
   HANDLE_ERROR( hipEventSynchronize( stop ) );
   float   t;
   HANDLE_ERROR( hipEventElapsedTime( &t, start, stop ) );
   HANDLE_ERROR( hipEventDestroy(start));
   HANDLE_ERROR( hipEventDestroy(stop));
   

   fp = fopen(oname, "w");
   fprintf(fp, "Beg. distance: %f; Steps: %d; Time to generate:  %3.1f ms\n", 
            b, l, t);
   Write(K, V, fp);
   fclose(fp);

   free(r);
   free(v);
   free(K);
   free(V);
//   hipUnbindTexture(tex_r);
//   hipUnbindTexture(tex_v);
//   hipUnbindTexture(tex_a);
//   hipFree(dev_an);
}
